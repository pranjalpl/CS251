
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>



#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);


__global__ void XOR(int *dA,int num, int offset){ 
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(((i+offset) < num) && ((i%(2*offset)) == 0)){	
	  dA[i] = dA[i]^dA[i+offset];
      }
}

int main(int argc, char **argv){
    int num,ctr,blocks;
    int seed;
    int offset = 1;
    num = atoi(argv[1]);
    seed = atoi(argv[2]);
    
    int *hA;
  
    int *dA; 
   
    int size = num * sizeof(int); 

    /*Allocate memory on the host (CPU) */

    hA = (int *) malloc(size);
    if(!hA){
          perror("malloc");
          exit(-1);
    }
    
    /*Initialize hA*/

    srand(seed);
    for(ctr=0; ctr < num; ++ctr){
	    hA[ctr] = random();
    }
    
   /*Allocate memory on the device (GPU) */

    hipMalloc(&dA,  size);
    CUDA_ERROR_EXIT("hipMalloc");
   
    /*Copy hA --> dA */
    
    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("memcpy1");
    
    
     /*Invoke the kernel*/
    blocks = num /1024;
    
    if(num % 1024)
           ++blocks;
    
    for(offset = 1; offset < num; offset*=2){
	    XOR<<<blocks, 1024>>>(dA,num,offset);
	    CUDA_ERROR_EXIT("kernel invocation");
    }
   
    printf("kernel successful\n"); 

    /*Copy back results*/
    hipMemcpy(hA, dA, size, hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");

    printf("%d\n", hA[0]);
    
    free(hA);  
    hipFree(dA);
}
